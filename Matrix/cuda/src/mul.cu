#include "hip/hip_runtime.h"
/********************************************

  Cuda Multiplication (actually +/- too)

  Taylor Shin

  July 17th 2017

*********************************************/

#include "mul.h"

#include <stdexcept>

/********************************************
  Cuda kernels
*********************************************/
template<typename T>
__global__
void add_kernel(void** ka, void** kb, void** kc)
{
  return;
}

template<typename T>
__global__
void add_sc_kernel(void** ka, void* kb, void** kc)
{
  return;
}

template<typename T>
__global__
void sub_kernel(void** ka, void** kb, void** kc)
{
  return;
}

template<typename T>
__global__
void sub_sc_kernel(void** ka, void* kb, void** kc)
{
  return;
}

template<typename T>
__global__
void mul_kernel(void** ka, void** kb, void** kc)
{
  return;
}

template<typename T>
__global__
void mul_sc_kernel(void** ka, void* kb, void** kc)
{
  return;
}

/********************************************
  Cuda Wrappers - Assume everything's malloced data
*********************************************/
#define NULL_CHECK(A) \
  if (!A) throw std::string("Can't calculate NULL pointer!!");

/* TODO: Gotta implement some wrapper here to reduce tediousness */

/* Cuda wrappers  */
template <typename T>
T* AddCuda(T* a, T* b, size_t r, size_t c)
{
  NULL_CHECK((a||b))
  
  size_t memsize = r*c*sizeof(T);
  
  T* res = (T*)malloc(memsize);
  int block_size, min_grid_size, grid_size;
  
  void** a_vec; hipMalloc((void**)&a_vec, memsize);
  void** b_vec; hipMalloc((void**)&b_vec, memsize);
  void** res_vec; hipMalloc((void**)&res_vec, memsize);
  hipMemcpy(a_vec, a, memsize, hipMemcpyHostToDevice);
  hipMemcpy(b_vec, b, memsize, hipMemcpyHostToDevice);
  
  hipOccupancyMaxPotentialBlockSize(
    &min_grid_size, &block_size, add_kernel<T>, 0, memsize);
  
  grid_size = (memsize+block_size-1)/block_size;
  
  add_kernel<T><<<grid_size,block_size>>>(a_vec, b_vec, res_vec);
  
  hipMemcpy((void**)&res, res_vec, memsize, hipMemcpyDeviceToHost);
  
  return res;
}

template <typename T>
T* AddScCuda(T* a, const T& sc, size_t r, size_t c)
{
  NULL_CHECK(a);
  
  size_t memsize = r*c*sizeof(T);
  
  T* res = (T*)malloc(memsize);
  int block_size, min_grid_size, grid_size;
  
  void** a_vec; hipMalloc((void**)&a_vec, memsize);
  void** sc_vec; hipMalloc((void**)&sc_vec, sizeof(T));
  void** res_vec; hipMalloc((void**)&res_vec, memsize);
  hipMemcpy(a_vec, a, memsize, hipMemcpyHostToDevice);
  hipMemcpy(sc_vec, &sc, sizeof(T), hipMemcpyHostToDevice);
  
  hipOccupancyMaxPotentialBlockSize(
    &min_grid_size, &block_size, add_sc_kernel<T>, 0, memsize);
  
  grid_size = (memsize+block_size-1)/block_size;
  
  add_sc_kernel<T><<<grid_size,block_size>>>(a_vec, *sc_vec, res_vec);
  
  hipMemcpy((void**)&res, res_vec, memsize, hipMemcpyDeviceToHost);
  
  return res;
}

template <typename T>
T* SubCuda(T* a, T* b, size_t r, size_t c)
{
}

template <typename T>
T* SubScCuda(T* a, const T& sc, size_t r, size_t c)
{
}

template <typename T>
T* MulCuda(T* a, T* b, size_t r, size_t c)
{
}

template <typename T>
T* MulScCuda(T* a, const T& sc, size_t r, size_t c)
{
}





/* Template instantiations */
ADD_CUDA_TEMPL(int)
ADD_CUDA_TEMPL(float)
ADD_CUDA_TEMPL(double)
ADD_CUDA_TEMPL(int64_t)