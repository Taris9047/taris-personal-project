#include "hip/hip_runtime.h"
/********************************************

  Cuda Multiplication (actually +/- too)

  Taylor Shin

  July 17th 2017

*********************************************/

#include "mul.h"

#include <stdexcept>

/********************************************
  Cuda kernels
*********************************************/
template<typename T>
__global__
void add_kernel(void* ka, void* kb, void* kc, size_t r, size_t c)
{
  auto n = r*c;
  T* a_data = (T*)ka;
  T* b_data = (T*)kb;
  T* c_data = (T*)kc;
  for (auto i=0; i<n; ++i) {
    c_data[i] = a_data[i] + b_data[i];
  }
  
  return;
}

template<typename T>
__global__
void add_sc_kernel(void* ka, void* kb, void* kc, size_t r, size_t c)
{
  return;
}

template<typename T>
__global__
void sub_kernel(void* ka, void* kb, void* kc, size_t r, size_t c)
{
  return;
}

template<typename T>
__global__
void sub_sc_kernel(void* ka, void* kb, void* kc, size_t r, size_t c)
{
  return;
}

template<typename T>
__global__
void mul_kernel(void* ka, void* kb, void* kc, size_t r, size_t c)
{
  return;
}

template<typename T>
__global__
void mul_sc_kernel(void* ka, void* kb, void* kc, size_t r, size_t c)
{
  return;
}


/********************************************
  Some typedefs for Cuda Kernels
*********************************************/
template <typename T>
using MtoMKernel = void (*)(void*, void*, void*, size_t, size_t);
template <typename T>
using MtoScKernel = void (*)(void*, void*, void*, size_t, size_t);

/********************************************
  Cuda Wrappers - Assume everything's malloced data
*********************************************/
#define NULL_CHECK(A) \
  if (!A) throw std::string("Can't calculate NULL pointer!!");

/* Static functions */
/* Prepares data to GPU and preforms given Kernel operation */
/* Matrix To Matrix */
template <typename T>
T* RK_MatMat(T* a, T* b, size_t r, size_t c, MtoMKernel<T> KERNEL_FUNC)
{
  NULL_CHECK((a||b)) 
  size_t memsize = r*c*sizeof(T); 
  
  T* res = (T*)malloc(memsize); 
  int block_size, min_grid_size, grid_size; 

  void* a_vec; hipMalloc((void**)&a_vec, memsize); 
  void* b_vec; hipMalloc((void**)&b_vec, memsize); 
  void* res_vec; hipMalloc((void**)&res_vec, memsize); 
  hipMemcpy(a_vec, a, memsize, hipMemcpyHostToDevice); 
  hipMemcpy(b_vec, b, memsize, hipMemcpyHostToDevice); 

  hipOccupancyMaxPotentialBlockSize(
    &min_grid_size, &block_size, add_kernel<T>, 0, memsize); 
  
  grid_size = (memsize+block_size-1)/block_size; 
  
  KERNEL_FUNC<<<grid_size,block_size>>>(a_vec, b_vec, res_vec, r, c); 
  
  hipMemcpy((void*)res, res_vec, memsize, hipMemcpyDeviceToHost); 
  
  hipFree(a_vec); hipFree(b_vec); hipFree(res_vec);
  
  return res;
}

/* Matrix To Scalar */
template <typename T>
T* RK_MatSc(T* a, const T& sc, size_t r, size_t c, MtoScKernel<T> KERNEL_FUNC)
{
  NULL_CHECK(a);
  
  size_t memsize = r*c*sizeof(T);
  
  T* res = (T*)malloc(memsize);
  int block_size, min_grid_size, grid_size;
  
  void* a_vec; hipMalloc((void**)&a_vec, memsize);
  void* sc_vec; hipMalloc((void**)&sc_vec, sizeof(T));
  void* res_vec; hipMalloc((void**)&res_vec, memsize);
  hipMemcpy(a_vec, a, memsize, hipMemcpyHostToDevice);
  hipMemcpy(sc_vec, &sc, sizeof(T), hipMemcpyHostToDevice);
  
  hipOccupancyMaxPotentialBlockSize(
    &min_grid_size, &block_size, add_sc_kernel<T>, 0, memsize);
  
  grid_size = (memsize+block_size-1)/block_size;
  
  KERNEL_FUNC<<<grid_size,block_size>>>(a_vec, sc_vec, res_vec, r, c);
  
  hipMemcpy((void*)res, res_vec, memsize, hipMemcpyDeviceToHost);
  
  hipFree(a_vec); hipFree(sc_vec); hipFree(res_vec);
  
  return res;
}

/* Cuda wrappers  */
template <typename T>
T* AddCuda(T* a, T* b, size_t r, size_t c)
{
  return RK_MatMat<T>(a,b,r,c,&add_kernel<T>);
}

template <typename T>
T* AddScCuda(T* a, const T& sc, size_t r, size_t c)
{
  return RK_MatSc<T>(a,sc,r,c,&add_sc_kernel<T>);
}

template <typename T>
T* SubCuda(T* a, T* b, size_t r, size_t c)
{
  return RK_MatMat<T>(a,b,r,c,&sub_kernel<T>);
}

template <typename T>
T* SubScCuda(T* a, const T& sc, size_t r, size_t c)
{
  return RK_MatSc<T>(a,sc,r,c,&sub_sc_kernel<T>);
}

template <typename T>
T* MulCuda(T* a, T* b, size_t r, size_t c)
{
  return RK_MatMat<T>(a,b,r,c,&mul_kernel<T>);
}

template <typename T>
T* MulScCuda(T* a, const T& sc, size_t r, size_t c)
{
  return RK_MatSc<T>(a,sc,r,c,&mul_sc_kernel<T>);
}





/* Template instantiations */
ADD_CUDA_TEMPL(int)
ADD_CUDA_TEMPL(float)
ADD_CUDA_TEMPL(double)
ADD_CUDA_TEMPL(int64_t)