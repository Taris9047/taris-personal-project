#include "hip/hip_runtime.h"
/********************************************

  Cuda Multiplication (actually +/- too)

  Taylor Shin

  July 17th 2017

*********************************************/

#include "mul.h"

#include <stdexcept>

/********************************************
  Cuda kernels
*********************************************/
template<typename T>
__global__
void add_kernel(void* ka, void* kb, void* kc, size_t r, size_t c)
{
  auto n = r*c;
  T* a_data = (T*)ka;
  T* b_data = (T*)kb;
  T* c_data = (T*)kc;
  for (auto i=0; i<n; ++i) {
    c_data[i] = a_data[i] + b_data[i];
  }
  
  return;
}

template<typename T>
__global__
void add_sc_kernel(void* ka, void* kb, void* kc, size_t r, size_t c)
{
  auto n = r*c;
  T* a_data = (T*)ka;
  T* b_data = (T*)kb;
  T* c_data = (T*)kc;
  for (auto i=0; i<n; ++i) {
    c_data[i] = a_data[i] + (*b_data);
  }

  return;
}

template<typename T>
__global__
void sub_kernel(void* ka, void* kb, void* kc, size_t r, size_t c)
{
  auto n = r*c;
  T* a_data = (T*)ka;
  T* b_data = (T*)kb;
  T* c_data = (T*)kc;
  for (auto i=0; i<n; ++i) {
    c_data[i] = a_data[i] - b_data[i];
  }
  
  return;
}

template<typename T>
__global__
void sub_sc_kernel(void* ka, void* kb, void* kc, size_t r, size_t c)
{
  auto n = r*c;
  T* a_data = (T*)ka;
  T* b_data = (T*)kb;
  T* c_data = (T*)kc;
  for (auto i=0; i<n; ++i) {
    c_data[i] = a_data[i] - (*b_data);
  }

  return;
}

/* TODO: this part is kinda wrong */
template<typename T>
__global__
void mul_kernel(void* ka, void* kb, void* kc, size_t a_r, size_t a_c, size_t b_r, size_t b_c)
{
  T* a_data = (T*)ka;
  T* b_data = (T*)kb;
  T* c_data = (T*)kc;
  
  T tmp;
  for (auto i=0; i<a_r; ++i) {
    for (auto j=0; j<a_c; ++j) {
      tmp = T();
      for (auto k=0; k<b_r; ++k) {
        tmp += a_data[i*a_r+k]*b_data[k*b_r+j];
      } /* for (k=0; k<b_r; ++k) */
      c_data[i*a_r+j] = tmp;
    } /* for (j=0; j<a_c; ++j) */
  } /* for (i=0; i<a_r; ++i) */
  
  return;
}

template<typename T>
__global__
void mul_sc_kernel(void* ka, void* kb, void* kc, size_t r, size_t c)
{
  auto n = r*c;
  T* a_data = (T*)ka;
  T* b_data = (T*)kb;
  T* c_data = (T*)kc;
  for (auto i=0; i<n; ++i) {
    c_data[i] = a_data[i] * (*b_data);
  }

  return;
}


/********************************************
  Some typedefs for Cuda Kernels
*********************************************/
template <typename T>
using MtoMKernel = void (*)(void*, void*, void*, size_t, size_t);
template <typename T>
using MtoScKernel = void (*)(void*, void*, void*, size_t, size_t);

/********************************************
  Cuda Wrappers - Assume everything's malloced data
*********************************************/
#define NULL_CHECK(A) \
  if (!A) throw std::string("Can't calculate NULL pointer!!");

/* Static functions */
/* Prepares data to GPU and preforms given Kernel operation */
/* Matrix To Matrix */
template <typename T>
T* RK_MatMat(T* a, T* b, size_t r, size_t c, MtoMKernel<T> KERNEL_FUNC)
{
  NULL_CHECK((a||b)) 
  size_t memsize = r*c*sizeof(T); 
  
  T* res = (T*)malloc(memsize); 
  int block_size, min_grid_size, grid_size; 

  void* a_vec; hipMalloc((void**)&a_vec, memsize); 
  void* b_vec; hipMalloc((void**)&b_vec, memsize); 
  void* res_vec; hipMalloc((void**)&res_vec, memsize); 
  hipMemcpy(a_vec, a, memsize, hipMemcpyHostToDevice); 
  hipMemcpy(b_vec, b, memsize, hipMemcpyHostToDevice); 

  hipOccupancyMaxPotentialBlockSize(
    &min_grid_size, &block_size, add_kernel<T>, 0, memsize); 
  
  grid_size = (memsize+block_size-1)/block_size; 
  
  KERNEL_FUNC<<<grid_size,block_size>>>(a_vec, b_vec, res_vec, r, c); 
  
  hipMemcpy((void*)res, res_vec, memsize, hipMemcpyDeviceToHost); 
  
  hipFree(a_vec); hipFree(b_vec); hipFree(res_vec);
  
  return res;
}

/* Matrix To Scalar */
template <typename T>
T* RK_MatSc(T* a, const T& sc, size_t r, size_t c, MtoScKernel<T> KERNEL_FUNC)
{
  NULL_CHECK(a);
  
  size_t memsize = r*c*sizeof(T);
  
  T* res = (T*)malloc(memsize);
  int block_size, min_grid_size, grid_size;
  
  void* a_vec; hipMalloc((void**)&a_vec, memsize);
  void* sc_vec; hipMalloc((void**)&sc_vec, sizeof(T));
  void* res_vec; hipMalloc((void**)&res_vec, memsize);
  hipMemcpy(a_vec, a, memsize, hipMemcpyHostToDevice);
  hipMemcpy(sc_vec, &sc, sizeof(T), hipMemcpyHostToDevice);
  
  hipOccupancyMaxPotentialBlockSize(
    &min_grid_size, &block_size, add_sc_kernel<T>, 0, memsize);
  
  grid_size = (memsize+block_size-1)/block_size;
  
  KERNEL_FUNC<<<grid_size,block_size>>>(a_vec, sc_vec, res_vec, r, c);
  
  hipMemcpy((void*)res, res_vec, memsize, hipMemcpyDeviceToHost);
  
  hipFree(a_vec); hipFree(sc_vec); hipFree(res_vec);
  
  return res;
}

/* Matrix to Matrix multiplication */
template <typename T>
T* RK_MatMatMul(T* a, T* b, size_t a_r, size_t a_c, size_t b_r, size_t b_c)
{
  NULL_CHECK((a||b)) 
  size_t a_memsize = a_r*a_c*sizeof(T);
  size_t b_memsize = b_r*b_c*sizeof(T);
  size_t c_memsize = a_r*b_c*sizeof(T);
  
  T* res = (T*)malloc(c_memsize); 
  int block_size, min_grid_size, grid_size; 

  void* a_vec; hipMalloc((void**)&a_vec, a_memsize); 
  void* b_vec; hipMalloc((void**)&b_vec, b_memsize); 
  void* res_vec; hipMalloc((void**)&res_vec, c_memsize); 
  hipMemcpy(a_vec, a, a_memsize, hipMemcpyHostToDevice); 
  hipMemcpy(b_vec, b, b_memsize, hipMemcpyHostToDevice); 

  hipOccupancyMaxPotentialBlockSize(
    &min_grid_size, &block_size, add_kernel<T>, 0, c_memsize); 
  
  grid_size = (c_memsize+block_size-1)/block_size; 
  
  mul_kernel<T><<<grid_size,block_size>>>(a_vec, b_vec, res_vec, a_r, a_c, b_r, b_c); 
  
  hipMemcpy((void*)res, res_vec, c_memsize, hipMemcpyDeviceToHost); 
  
  hipFree(a_vec); hipFree(b_vec); hipFree(res_vec);
  
  return res;
}

/* Cuda wrappers  */
template <typename T>
T* AddCuda(T* a, T* b, size_t r, size_t c)
{
  return RK_MatMat<T>(a,b,r,c,&add_kernel<T>);
}

template <typename T>
T* AddScCuda(T* a, const T& sc, size_t r, size_t c)
{
  return RK_MatSc<T>(a,sc,r,c,&add_sc_kernel<T>);
}

template <typename T>
T* SubCuda(T* a, T* b, size_t r, size_t c)
{
  return RK_MatMat<T>(a,b,r,c,&sub_kernel<T>);
}

template <typename T>
T* SubScCuda(T* a, const T& sc, size_t r, size_t c)
{
  return RK_MatSc<T>(a,sc,r,c,&sub_sc_kernel<T>);
}

template <typename T>
T* MulCuda(T* a, T* b, size_t a_r, size_t a_c, size_t b_r, size_t b_c)
{
  return RK_MatMatMul<T>(a,b,a_r,a_c,b_r,b_c);
}

template <typename T>
T* MulScCuda(T* a, const T& sc, size_t r, size_t c)
{
  return RK_MatSc<T>(a,sc,r,c,&mul_sc_kernel<T>);
}





/* Template instantiations */
ADD_CUDA_TEMPL(int)
ADD_CUDA_TEMPL(float)
ADD_CUDA_TEMPL(double)
ADD_CUDA_TEMPL(int64_t)