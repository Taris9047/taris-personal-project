#include "hip/hip_runtime.h"
/********************************************

  Cuda Multiplication (actually +/- too)

  Taylor Shin

  July 17th 2017

*********************************************/

#include "mul.h"

#include <stdexcept>

/********************************************
  Block assignment tool
*********************************************/
#define MAX_THREADS_PER_BLOCK_SQRT 8
static inline void assign_blocks_mul(size_t rows, size_t cols, dim3* blocks, dim3* thr_per_block)
{
  size_t r_threads, c_threads;
  if (rows < MAX_THREADS_PER_BLOCK_SQRT) r_threads = rows/2;
  else r_threads = MAX_THREADS_PER_BLOCK_SQRT;
  if (cols < MAX_THREADS_PER_BLOCK_SQRT) c_threads = cols/2;
  else c_threads = MAX_THREADS_PER_BLOCK_SQRT;
  if (!r_threads) r_threads = 1;
  if (!c_threads) c_threads = 1;
  
  dim3 tpb(r_threads, c_threads);
  dim3 nb(rows/tpb.x, cols/tpb.y);
  
  (*blocks) = nb;
  (*thr_per_block) = tpb;
}

/********************************************
  Cuda kernels
*********************************************/
template<typename T>
__global__
void add_kernel(void* ka, void* kb, void* kc, size_t r, size_t c)
{
  auto n = r*c;
  T* a_data = (T*)ka;
  T* b_data = (T*)kb;
  T* c_data = (T*)kc;
  auto index = blockIdx.x * blockDim.x + threadIdx.x;
  auto stride = blockDim.x * gridDim.x;
  for (auto i=index; i<n; i+=stride)
    c_data[i] = a_data[i] + b_data[i];
  
  return;
}

template<typename T>
__global__
void add_sc_kernel(void* ka, void* kb, void* kc, size_t r, size_t c)
{
  auto n = r*c;
  T* a_data = (T*)ka;
  T* b_data = (T*)kb;
  T* c_data = (T*)kc;
  auto index = blockIdx.x * blockDim.x + threadIdx.x;
  auto stride = blockDim.x * gridDim.x;
  for (auto i=index; i<n; i+=stride) {
    c_data[i] = a_data[i] + (*b_data);
  }

  return;
}

template<typename T>
__global__
void sub_kernel(void* ka, void* kb, void* kc, size_t r, size_t c)
{
  auto n = r*c;
  T* a_data = (T*)ka;
  T* b_data = (T*)kb;
  T* c_data = (T*)kc;
  auto index = blockIdx.x * blockDim.x + threadIdx.x;
  auto stride = blockDim.x * gridDim.x;
  for (auto i=index; i<n; i+=stride) {
    c_data[i] = a_data[i] - b_data[i];
  }
  
  return;
}

template<typename T>
__global__
void sub_sc_kernel(void* ka, void* kb, void* kc, size_t r, size_t c)
{
  auto n = r*c;
  T* a_data = (T*)ka;
  T* b_data = (T*)kb;
  T* c_data = (T*)kc;
  auto index = blockIdx.x * blockDim.x + threadIdx.x;
  auto stride = blockDim.x * gridDim.x;
  for (auto i=index; i<n; i+=stride) {
    c_data[i] = a_data[i] - (*b_data);
  }

  return;
}

/* TODO: this part is kinda wrong */
template<typename T>
__global__
void mul_kernel(void* ka, void* kb, void* kc, size_t a_r, size_t a_c, size_t b_r, size_t b_c)
{
  T* a_data = (T*)ka;
  T* b_data = (T*)kb;
  T* c_data = (T*)kc;
  
  T tmp;
  auto index_r = blockIdx.x * blockDim.x + threadIdx.x;
  auto stride_r = blockDim.x * gridDim.x;
  for (auto i=index_r; i<a_r; i+=stride_r) {
    auto index_c = blockIdx.y * blockDim.y + threadIdx.y;
    auto stride_c = blockDim.y * gridDim.y;
    for (auto j=index_c; j<b_c; j+=stride_c) {
      tmp = T();
      auto index_x = blockIdx.z * blockDim.z + threadIdx.z;
      auto stride_x = blockDim.z * gridDim.z;
      for (auto k=index_x; k<b_r; k+=stride_x) {
      //for (auto k=0; k<b_r; ++k) {
        tmp += a_data[i*a_c+k]*b_data[k*b_c+j];
      } /* for (k=0; k<b_r; ++k) */
      c_data[i*b_c+j] = tmp;
    } /* for (j=0; j<a_c; ++j) */
  } /* for (i=0; i<a_r; ++i) */

  return;
}

template<typename T>
__global__
void mul_sc_kernel(void* ka, void* kb, void* kc, size_t r, size_t c)
{
  auto n = r*c;
  T* a_data = (T*)ka;
  T* b_data = (T*)kb;
  T* c_data = (T*)kc;
  auto index = blockIdx.x * blockDim.x + threadIdx.x;
  auto stride = blockDim.x * gridDim.x;
  for (auto i=index; i<n; i+=stride) {
    c_data[i] = a_data[i] * (*b_data);
  }

  return;
}


/********************************************
  Some typedefs for Cuda Kernels
*********************************************/
template <typename T>
using MtoMKernel = void (*)(void*, void*, void*, size_t, size_t);
template <typename T>
using MtoScKernel = void (*)(void*, void*, void*, size_t, size_t);

/********************************************
  Cuda Wrappers - Assume everything's malloced data
*********************************************/
#define NULL_CHECK(A) \
  if (!A) throw std::string("Can't calculate NULL pointer!!");

/* Static functions */
/* Prepares data to GPU and preforms given Kernel operation */
/* Matrix To Matrix */
template <typename T>
T* RK_MatMat(T* a, T* b, size_t r, size_t c, MtoMKernel<T> KERNEL_FUNC)
{
  NULL_CHECK((a||b)) 
  size_t memsize = r*c*sizeof(T); 
  
  T* res = (T*)malloc(memsize);
  int block_size, min_grid_size, grid_size; 

  void* a_vec; hipMallocManaged((void**)&a_vec, memsize); 
  void* b_vec; hipMallocManaged((void**)&b_vec, memsize); 
  void* res_vec; hipMallocManaged((void**)&res_vec, memsize); 
  hipMemcpy(a_vec, a, memsize, hipMemcpyHostToDevice); 
  hipMemcpy(b_vec, b, memsize, hipMemcpyHostToDevice); 

  hipOccupancyMaxPotentialBlockSize(
    &min_grid_size, &block_size, KERNEL_FUNC, 0, memsize); 
  
  grid_size = (memsize+block_size-1)/block_size; 
  
  KERNEL_FUNC<<<grid_size,block_size>>>(a_vec, b_vec, res_vec, r, c); 
  
  hipMemcpy((void*)res, res_vec, memsize, hipMemcpyDeviceToHost); 
  
  hipFree(a_vec); hipFree(b_vec); hipFree(res_vec);
  
  return res;
}

/* Matrix To Scalar */
template <typename T>
T* RK_MatSc(T* a, const T& sc, size_t r, size_t c, MtoScKernel<T> KERNEL_FUNC)
{
  NULL_CHECK(a);
  
  size_t memsize = r*c*sizeof(T);
  
  T* res = (T*)malloc(memsize);
  int block_size, min_grid_size, grid_size;
  
  void* a_vec; hipMallocManaged((void**)&a_vec, memsize);
  void* sc_vec; hipMallocManaged((void**)&sc_vec, sizeof(T));
  void* res_vec; hipMallocManaged((void**)&res_vec, memsize);
  hipMemcpy(a_vec, a, memsize, hipMemcpyHostToDevice);
  hipMemcpy(sc_vec, &sc, sizeof(T), hipMemcpyHostToDevice);
  
  hipOccupancyMaxPotentialBlockSize(
    &min_grid_size, &block_size, KERNEL_FUNC, 0, memsize);
  
  grid_size = (memsize+block_size-1)/block_size;
  
  KERNEL_FUNC<<<grid_size,block_size>>>(a_vec, sc_vec, res_vec, r, c);
  
  hipMemcpy((void*)res, res_vec, memsize, hipMemcpyDeviceToHost);
  
  hipFree(a_vec); hipFree(sc_vec); hipFree(res_vec);
  
  return res;
}

/* Matrix to Matrix multiplication */
template <typename T>
T* RK_MatMatMul(T* a, T* b, size_t a_r, size_t a_c, size_t b_r, size_t b_c)
{
  NULL_CHECK((a||b)) 
  size_t a_memsize = a_r*a_c*sizeof(T);
  size_t b_memsize = b_r*b_c*sizeof(T);
  size_t c_memsize = a_r*b_c*sizeof(T);
  
  T* res = (T*)malloc(c_memsize); 

  void* a_vec; hipMallocManaged((void**)&a_vec, a_memsize); 
  void* b_vec; hipMallocManaged((void**)&b_vec, b_memsize); 
  void* res_vec; hipMallocManaged((void**)&res_vec, c_memsize); 
  hipMemcpy(a_vec, a, a_memsize, hipMemcpyHostToDevice); 
  hipMemcpy(b_vec, b, b_memsize, hipMemcpyHostToDevice); 

  // int block_size, min_grid_size, grid_size; 
  // hipOccupancyMaxPotentialBlockSize(
  //   &min_grid_size, &block_size, mul_kernel<T>, 0, c_memsize); 
  // 
  // grid_size = (c_memsize+block_size-1)/block_size;
  // mul_kernel<T><<<grid_size,block_size>>>(a_vec, b_vec, res_vec, a_r, a_c, b_r, b_c); 
  
  // dim3 threadsPerBlock(8, 8);
  // dim3 numBlocks(a_r/threadsPerBlock.x, b_c/threadsPerBlock.y);
  dim3 threadsPerBlock;
  dim3 numBlocks;
  
  assign_blocks_mul(a_r, b_c, &numBlocks, &threadsPerBlock);
  
  mul_kernel<T><<<numBlocks,threadsPerBlock>>>(a_vec, b_vec, res_vec, a_r, a_c, b_r, b_c); 
  
  hipMemcpy((void*)res, res_vec, c_memsize, hipMemcpyDeviceToHost); 
  
  hipFree(a_vec); hipFree(b_vec); hipFree(res_vec);
  
  return res;
}

/* Cuda wrappers  */
template <typename T>
T* AddCuda(T* a, T* b, size_t r, size_t c)
{
  return RK_MatMat<T>(a,b,r,c,&add_kernel<T>);
}

template <typename T>
T* AddScCuda(T* a, const T& sc, size_t r, size_t c)
{
  return RK_MatSc<T>(a,sc,r,c,&add_sc_kernel<T>);
}

template <typename T>
T* SubCuda(T* a, T* b, size_t r, size_t c)
{
  return RK_MatMat<T>(a,b,r,c,&sub_kernel<T>);
}

template <typename T>
T* SubScCuda(T* a, const T& sc, size_t r, size_t c)
{
  return RK_MatSc<T>(a,sc,r,c,&sub_sc_kernel<T>);
}

template <typename T>
T* MulCuda(T* a, T* b, size_t a_r, size_t a_c, size_t b_r, size_t b_c)
{
  return RK_MatMatMul<T>(a,b,a_r,a_c,b_r,b_c);
}

template <typename T>
T* MulScCuda(T* a, const T& sc, size_t r, size_t c)
{
  return RK_MatSc<T>(a,sc,r,c,&mul_sc_kernel<T>);
}





/* Template instantiations */
ADD_CUDA_TEMPL(int)
ADD_CUDA_TEMPL(float)
ADD_CUDA_TEMPL(double)
ADD_CUDA_TEMPL(int64_t)