#include "hip/hip_runtime.h"
/********************************************

  Cuda Multiplication (actually +/- too)

  Taylor Shin

  July 17th 2017

*********************************************/

#include "mul.h"

#include <stdexcept>

/********************************************
  Cuda kernels
*********************************************/
__global__
void add_kernel(void** ka, void** kb, void*** kc)
{
  return;
}

__global__
void add_sc_kernel(void** ka, void** kb, void*** kc)
{
  return;
}

__global__
void sub_kernel(void** ka, void** kb, void*** kc)
{
  return;
}

__global__
void sub_sc_kernel(void** ka, void** kb, void*** kc)
{
  return;
}

__global__
void mul_kernel(void** ka, void** kb, void*** kc)
{
  return;
}

__global__
void mul_sc_kernel(void** ka, void** kb, void*** kc)
{
  return;
}

/********************************************
  Cuda Wrappers - Assume everything's malloced data
*********************************************/
/* Double precision */
double** AddCudaDBL(double** a, double** b, size_t rows, size_t cols)
{
  if (!a||!b) throw std::string("Can't calculate NULL pointer!!");

  double** c;
  
//  add_kernel<<<1,128>>>(kernel_a, kernel_b, &kernel_c);

  return c;
}

double** AddScCudaDBL(double** a, double s, size_t rows, size_t cols)
{
}

double** SubCudaDBL(double** a, double** b, size_t rows, size_t cols)
{
}

double** SubScCudaDBL(double** a, double s, size_t rows, size_t cols)
{
}


double** MulCudaDBL(double** a, double** b, size_t rows, size_t cols)
{
}

double** MulScCudaDBL(double** a, double s, size_t rows, size_t cols)
{
}


/* Floating point */
float** AddCudaFLT(float**, float**, size_t rows, size_t cols)
{
}
float** AddScCudaFLT(float**, float, size_t rows, size_t cols)
{
}
float** SubCudaFLT(float**, float**, size_t rows, size_t cols)
{
}
float** SubScCudaFLT(float**, float, size_t rows, size_t cols)
{
}

float** MulCudaFLT(float**, float**, size_t rows, size_t cols)
{
}
float** MulScCudaFLT(float**, float, size_t rows, size_t cols)
{
}


/* Long integer */
int64_t** AddCudaLong(int64_t**, int64_t**, size_t rows, size_t cols)
{
}
int64_t** AddScCudaLong(int64_t**, int64_t, size_t rows, size_t cols)
{
}
int64_t** SubCudaLong(int64_t**, int64_t**, size_t rows, size_t cols)
{
}
int64_t** SubScCudaLong(int64_t**, int64_t, size_t rows, size_t cols)
{
}

int64_t** MulCudaLong(int64_t**, int64_t**, size_t rows, size_t cols)
{
}
int64_t** MulScCudaLong(int64_t**, int64_t, size_t rows, size_t cols)
{
}


/* Integer */
int** AddCudaInt(int**, int**, size_t rows, size_t cols)
{
}
int** AddScCudaInt(int**, int, size_t rows, size_t cols)
{
}
int** SubCudaInt(int**, int**, size_t rows, size_t cols)
{
}
int** SubScCudaInt(int**, int, size_t rows, size_t cols)
{
}

int** MulCudaInt(int**, int**, size_t rows, size_t cols)
{
}
int** MulScCudaInt(int**, int, size_t rows, size_t cols)
{
}