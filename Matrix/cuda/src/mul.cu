#include "hip/hip_runtime.h"
/********************************************

  Cuda Multiplication (actually +/- too)

  Taylor Shin

  July 17th 2017

*********************************************/

#include "mul.h"

#include <stdexcept>

/********************************************
  Cuda kernels
*********************************************/
__global__
void add_kernel(void** ka, void** kb, void*** kc)
{
  return;
}

__global__
void add_sc_kernel(void** ka, void** kb, void*** kc)
{
  return;
}

__global__
void sub_kernel(void** ka, void** kb, void*** kc)
{
  return;
}

__global__
void sub_sc_kernel(void** ka, void** kb, void*** kc)
{
  return;
}

__global__
void mul_kernel(void** ka, void** kb, void*** kc)
{
  return;
}

__global__
void mul_sc_kernel(void** ka, void** kb, void*** kc)
{
  return;
}

/********************************************
  Cuda Wrappers - Assume everything's malloced data
*********************************************/
double** AddCuda(double** a, double** b, size_t rows, size_t cols)
{
  if (!a||!b) throw std::string("Can't calculate NULL pointer!!");

  double** c;
  
//  add_kernel<<<1,128>>>(kernel_a, kernel_b, &kernel_c);

  return c;
}

double** AddScCuda(double** a, double s, size_t rows, size_t cols)
{
}

double** SubCuda(double** a, double** b, size_t rows, size_t cols)
{
}

double** SubScCuda(double** a, double s, size_t rows, size_t cols)
{
}


double** MulCuda(double** a, double** b, size_t rows, size_t cols)
{
}

double** MulScCuda(double** a, double s, size_t rows, size_t cols)
{
}
