#include "hip/hip_runtime.h"
/********************************************

  Cuda Multiplication (actually +/- too)

  Taylor Shin

  July 17th 2017

*********************************************/

#include "mul.h"

#include <stdexcept>

/********************************************
  Cuda kernels
*********************************************/
__global__
void add_kernel_int(void** ka, void** kb, void** kc)
{
  return;
}

__global__
void add_sc_kernel_int(void** ka, void* kb, void** kc)
{
  return;
}

__global__
void sub_kernel_int(void** ka, void** kb, void** kc)
{
  return;
}

__global__
void sub_sc_kernel_int(void** ka, void* kb, void** kc)
{
  return;
}

__global__
void mul_kernel_int(void** ka, void** kb, void** kc)
{
  return;
}

__global__
void mul_sc_kernel_int(void** ka, void* kb, void** kc)
{
  return;
}

/********************************************
  Cuda Wrappers - Assume everything's malloced data
*********************************************/
#define NULL_CHECK(A) \
  if (!A) throw std::string("Can't calculate NULL pointer!!");

/* Double precision */
double* AddCudaDBL(double* a, double* b, size_t rows, size_t cols)
{
  NULL_CHECK((a||b))

  double* c;
  


  return c;
}

double* AddScCudaDBL(double* a, double s, size_t rows, size_t cols)
{
  NULL_CHECK(a)
}

double* SubCudaDBL(double* a, double* b, size_t rows, size_t cols)
{
  NULL_CHECK((a||b))
}

double* SubScCudaDBL(double* a, double s, size_t rows, size_t cols)
{
  NULL_CHECK(a)
}


double* MulCudaDBL(double* a, double* b, size_t rows, size_t cols)
{
  NULL_CHECK((a||b))
}

double* MulScCudaDBL(double* a, double s, size_t rows, size_t cols)
{
  NULL_CHECK(a)
}


/* Floating point */
float* AddCudaFLT(float* a, float* b, size_t rows, size_t cols)
{
}
float* AddScCudaFLT(float* a, float sc, size_t rows, size_t cols)
{
}
float* SubCudaFLT(float* a, float* b, size_t rows, size_t cols)
{
}
float* SubScCudaFLT(float* a, float sc, size_t rows, size_t cols)
{
}

float* MulCudaFLT(float* a, float* b, size_t rows, size_t cols)
{
}
float* MulScCudaFLT(float* a, float sc, size_t rows, size_t cols)
{
}


/* Long integer */
int64_t* AddCudaLong(int64_t* a, int64_t* b, size_t rows, size_t cols)
{
}
int64_t* AddScCudaLong(int64_t* a, int64_t sc, size_t rows, size_t cols)
{
}
int64_t* SubCudaLong(int64_t* a, int64_t* b, size_t rows, size_t cols)
{
}
int64_t* SubScCudaLong(int64_t* a, int64_t sc, size_t rows, size_t cols)
{
}

int64_t* MulCudaLong(int64_t* a, int64_t* b, size_t rows, size_t cols)
{
}
int64_t* MulScCudaLong(int64_t* a, int64_t sc, size_t rows, size_t cols)
{
}


/* Integer */
int* AddCudaInt(int* a, int* b, size_t rows, size_t cols)
{
  NULL_CHECK((a||b))
  
  size_t memsize = rows*cols*sizeof(int);
  
  int* c = (int*)malloc(memsize);
  int block_size;
  int min_grid_size, grid_size;
  
  void** a_vec; hipMalloc((void**)&a_vec, memsize);
  void** b_vec; hipMalloc((void**)&b_vec, memsize);
  void** c_vec; hipMalloc((void**)&c_vec, memsize);
  hipMemcpy(a_vec, a, memsize, hipMemcpyHostToDevice);
  hipMemcpy(b_vec, b, memsize, hipMemcpyHostToDevice);
  
  hipOccupancyMaxPotentialBlockSize(
    &min_grid_size, &block_size, add_kernel_int, 0, memsize);
  
  grid_size = (memsize+block_size-1)/block_size;
  
  add_kernel_int<<<grid_size,block_size>>>(a_vec, b_vec, c_vec);
  
  hipMemcpy((void**)&c, c_vec, memsize, hipMemcpyDeviceToHost);
  
  return c;
}
int* AddScCudaInt(int* a, int sc, size_t rows, size_t cols)
{
  NULL_CHECK(a)
  int* c;
  add_sc_kernel_int<<<128,1>>>((void**)&a, (void*)&sc, (void**)&c);
  return c;
}
int* SubCudaInt(int* a, int* b, size_t rows, size_t cols)
{
  NULL_CHECK((a||b))
}
int* SubScCudaInt(int* a, int sc, size_t rows, size_t cols)
{
  NULL_CHECK(a)
}

int* MulCudaInt(int* a, int* b, size_t rows, size_t cols)
{
  NULL_CHECK((a||b))
}
int* MulScCudaInt(int* a, int sc, size_t rows, size_t cols)
{
  NULL_CHECK(a)
}